#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define DataType float
#define TPB 128

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
    out[i] = in1[i] + in2[i];
}

void classic(int power, int S_seg){
    int inputLength = 2 ** power;

    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *resultRef;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;
    
    hostInput1 = (DataType*) malloc(inputLength*sizeof(DataType));
    hostInput2 = (DataType*) malloc(inputLength*sizeof(DataType));
    hostOutput = (DataType*) malloc(inputLength*sizeof(DataType));
    hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength*sizeof(DataType));

    srand(time(NULL));
    for(int i = 0; i < inputLength; i++){
        hostInput1[i] = (DataType) rand() / RAND_MAX;
        hostInput2[i] = (DataType) rand() / RAND_MAX;
    }

    double iStart = cpuSecond();

    hipMemcpy(deviceInput1, hostInput1, inputLength*sizeof(DataType), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength*sizeof(DataType), hipMemcpyHostToDevice);
    vecAdd<<<(inputLength+TPB-1)/TPB,TPB>>>(deviceInput1, deviceInput2, deviceOutput, 0);
    hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(DataType), hipMemcpyDeviceToHost);

    double diff = cpuSecond() - iStart;
    printf("Power %02d: %f\n", power, diff);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
}

void improved(int power, int S_seg){
    int inputLength = 2 ** power;

    int nStreams = 4;

    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;

    hipHostMalloc((void **)&hostInput1, inputLength*sizeof(DataType));
    hipHostMalloc((void **)&hostInput2, inputLength*sizeof(DataType));
    hipHostMalloc((void **)&hostOutput, inputLength*sizeof(DataType));
    hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
    hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
    hipMalloc(&deviceOutput, inputLength*sizeof(DataType));

    srand(time(NULL));
    for (size_t i = 0; i < inputLength; i++) {
        hostInput1[i] = (float)rand() / RAND_MAX;
        hostInput2[i] = (float)rand() / RAND_MAX;
    }
    
    double iStart = cpuSecond();

    int N_seg = (inputLength + S_seg - 1) / S_seg;

    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++)
    {
        hipStreamCreate(&stream[i]);
    }

    for (int i = 0; i < N_seg; i++)
    {
        int offset = i * S_seg;
        int idx = i % nStreams;
        hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], S_seg*sizeof(DataType), hipMemcpyHostToDevice, stream[idx]);
        hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], S_seg*sizeof(DataType), hipMemcpyHostToDevice, stream[idx]);
        vecAdd<<<S_seg/TPB, TPB, 0, stream[idx]>>>(deviceInput1, deviceInput2, deviceOutput, offset);
        hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], S_seg*sizeof(DataType), hipMemcpyDeviceToHost, stream[idx]);
    }
    
    for (int i = 0; i < nStreams; i++)
    {
        hipStreamDestroy(&stream[i]);
    }
    hipDeviceSynchronize()
    double diff = cpuSecond() - iStart;
    printf("Power %02d, S_seg %05d: %f\n", power, S_seg, diff);

    hipHostFree(hostInput1);
    hipHostFree(hostInput2);
    hipHostFree(hostOutput);
    hipFree(deviceInput1);
    hipFree(deviceInput1);
    hipFree(deviceOutput);
}

int main(int argc, char **argv) {

    int lengths[5] = {4, 8, 12, 16, 20};
    int S_seg = 2 ** 4;

    printf("CLassic:\n")
    for (int i = 0; i < 5; i++)
    {
        classic(lengths[i], S_seg);
    }
    printf("\n---\n")
    printf("Improved:")
    for (int i = 0; i < 5; i++)
    {
        improved(lengths[i], S_seg);
    }
}